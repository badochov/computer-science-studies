
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <unordered_map>
#include <vector>
#include <algorithm>

#include <vector>
#include <stdexcept>
#include <iostream>

struct CSR {
  std::vector<size_t> row_ptr;
  std::vector<size_t> col_ind;
};

namespace gpu {
constexpr size_t WARP_SIZE = 32;
constexpr size_t GROUPS_PER_BLOCK = 4;
constexpr size_t THREADS = WARP_SIZE * GROUPS_PER_BLOCK;
constexpr size_t BLOCKS = 256;
constexpr size_t MAX_K = 12;
constexpr size_t STACK_SIZE = MAX_K - 2; // Last frame and first frame are redundant.
constexpr size_t MAX_NEIGHBOURS = 1024;
constexpr unsigned long long MOD = 1e9;

__device__ const size_t FULL_MASK = 0xffffffff;

struct BitwiseAdjList {
  static constexpr size_t LIST_SIZE = MAX_NEIGHBOURS / sizeof(uint32_t) / 8;
  uint32_t list[LIST_SIZE];

  __device__ bool is_neighbour(unsigned int y) {
    return ((list[y / 32]) & (1u << (y % 32))) != 0;
  }

  __device__  int intersect(const BitwiseAdjList &other, BitwiseAdjList *result) {
    int localNeighbours = 0;
    for (unsigned int i = threadIdx.x; i < LIST_SIZE; i += blockDim.x) {
      result->list[i] = list[i] & other.list[i];
      localNeighbours += __popc(result->list[i]);
    }
    return localNeighbours;
  }

  __device__  int intersect(const BitwiseAdjList &other) {
    int localNeighbours = 0;
    for (unsigned int i = threadIdx.x; i < LIST_SIZE; i += blockDim.x) {
      auto el = list[i] & other.list[i];
      localNeighbours += __popc(el);
    }
    return localNeighbours;
  }

  __device__ int copy(const BitwiseAdjList &other) {
    int local_neighbours = 0;
    for (unsigned int i = threadIdx.x; i < LIST_SIZE; i += blockDim.x) {
      list[i] = other.list[i];
      local_neighbours += __popc(list[i]);
    }
    return local_neighbours;
  }
};

__device__ unsigned threadTotalIdx() {
  return threadIdx.x + threadIdx.y * blockDim.x;
}

// AdjMatrix stores adjacency matrix using binary encoding.
struct AdjMatrix {
  BitwiseAdjList matrix[MAX_NEIGHBOURS];
  size_t dirty_list_idx[MAX_NEIGHBOURS];

  __device__ void fill(const size_t *subgraph_vertices,
                       size_t subgraph_vertices_count,
                       const size_t *row_ptr,
                       const size_t *col_ind) {
    // Each threads processes different row.
    for (unsigned int remapV = threadTotalIdx(); remapV < subgraph_vertices_count; remapV += blockDim.x * blockDim.y) {
      size_t remapIdx = 0;
      size_t orgV = subgraph_vertices[remapV];
      uint32_t neighbours = 0;
      for (size_t i = row_ptr[orgV]; i < row_ptr[orgV + 1]; i++) {
        // Neighbours are always in ascending order.
        size_t neighbour = col_ind[i];
        while ((remapIdx < subgraph_vertices_count) && (neighbour > subgraph_vertices[remapIdx])) {
          remapIdx++;
          if (remapIdx % 32 == 0) {
            matrix[remapV].list[(remapIdx / 32) - 1] = neighbours;
            neighbours = 0;
          }
        }
        // End of possible neighbours;
        if (remapIdx >= subgraph_vertices_count) {
          break;
        }
        // Add neighbour.
        if (neighbour == subgraph_vertices[remapIdx]) {
          neighbours |= 1u << (remapIdx % 32);
        }
      }
      size_t last_idx = remapIdx / 32;
      matrix[remapV].list[last_idx] = neighbours;
      // Clean trailing zeroes.
      for (size_t idx = last_idx + 1; idx <= dirty_list_idx[remapV]; idx++) {
        matrix[remapV].list[idx] = 0;
      }
      dirty_list_idx[remapV] = last_idx;
    }
  }
};

__device__ void reduceBlockRes(unsigned long long blockRes[THREADS / 2][MAX_K - 2],
                               const unsigned long long localRes[MAX_K],
                               unsigned long k) {
// Process first row separately to save space.
  for (int i = 1; i < k - 1; i++) { // First cell is not processed.
    atomicAdd(&blockRes[threadTotalIdx() / 2][i - 1], localRes[i]);
  }
  __syncthreads();

  for (int diff = 2; diff < THREADS; diff *= 2) {
    if ((threadTotalIdx() % diff == 0) && (threadTotalIdx() % (diff * 2) != 0)) {
      for (int i = 0; i < k - 2; i++) { // First cell is not processed.
        atomicAdd(&blockRes[(threadTotalIdx() - diff) / 2][i], blockRes[threadTotalIdx() / 2][i]);
      }
    }
    __syncthreads();
  }
}

__device__ void reduceRes(unsigned long long int blockRes[THREADS / 2][MAX_K - 2],
                          const unsigned long long localRes[MAX_K],
                          unsigned long long int *out,
                          unsigned long k) {
  atomicAdd(&out[0], localRes[0]);
  for (size_t i = 1; i < k - 1; i++) {
    atomicAdd(&out[i], blockRes[0][i - 1] % MOD);
  }
}

__global__ void clique_kernel(size_t *row_ptr,
                              size_t row_ptr_size,
                              size_t *col_ind,
                              unsigned long k,
                              unsigned long long *out,
                              unsigned long long *counter,
                              AdjMatrix matrices[BLOCKS]) {
  bool isGroupLeader = threadIdx.x == 0;
  bool isBlockLeader = isGroupLeader && threadIdx.y == 0;
  unsigned int groupId = threadIdx.y;

  __shared__ unsigned long long idx;
  __shared__ unsigned long long blockCounter;
  __shared__ unsigned long long groupVertexIdx[GROUPS_PER_BLOCK];

  __shared__ BitwiseAdjList blockStack[GROUPS_PER_BLOCK][STACK_SIZE];

  BitwiseAdjList *stack = blockStack[groupId];
  AdjMatrix &adj = matrices[blockIdx.x];

  size_t stackListIdx[STACK_SIZE];

  size_t *subgraph_vertices;
  size_t subgraph_vertices_count;

  unsigned long long localRes[MAX_K - 1] = {};

  for (;;) {
    __syncthreads(); // Do not get another job until all groups have finished processing.
    if (isBlockLeader) {
      idx = atomicAdd(counter, 1);
      blockCounter = 0;
    }
    __syncthreads();

    if (idx + 1 >= row_ptr_size) {
      break;
    }

    subgraph_vertices = &col_ind[row_ptr[idx]];
    subgraph_vertices_count = row_ptr[idx + 1] - row_ptr[idx];
    localRes[0] += subgraph_vertices_count;
    localRes[0] %= MOD;

    if (k == 2) {
      continue;
    }

    // Vertices are remapped.
    adj.fill(subgraph_vertices, subgraph_vertices_count, row_ptr, col_ind);
    __syncthreads();

    // From now on the work is split between groups.
    for (;;) {
      __syncwarp();
      if (isGroupLeader) {
        groupVertexIdx[groupId] = atomicAdd(&blockCounter, 1);
      }
      __syncwarp();

      if (groupVertexIdx[groupId] >= subgraph_vertices_count) {
        break;
      }

      localRes[1] += stack[0].copy(adj.matrix[groupVertexIdx[groupId]]);
      localRes[1] %= MOD;
      if (k == 3) {
        continue;
      }
      stackListIdx[0] = 0;

//    𝑛𝑢𝑚𝐶𝑙𝑖𝑞𝑢𝑒𝑠 = 0
//    procedure 𝑡𝑟𝑎𝑣𝑒𝑟𝑠𝑒𝑆𝑢𝑏𝑡𝑟𝑒𝑒 (𝐺, 𝑘, ℓ, 𝐼)
//    for 𝑣 ∈ 𝐼
//      𝐼′ = 𝐼 ∩ 𝐴𝑑𝑗𝐺 (𝑣)
//      if ℓ + 1 == 𝑘
//        𝑛𝑢𝑚𝐶𝑙𝑖𝑞𝑢𝑒𝑠 + = |𝐼′|
//      else if |𝐼′| > 0
//        𝑡𝑟𝑎𝑣𝑒𝑟𝑠𝑒𝑆𝑢𝑏𝑡𝑟𝑒𝑒 (𝐺, 𝑘, ℓ + 1, 𝐼′)

      unsigned long long level = 2;
      while (level != 1) {
        bool recurse = false;
        const bool lastLevel = level == k;
        BitwiseAdjList &frame = stack[level - 2];
        size_t &frame_list_idx = stackListIdx[level - 2];

        while (frame_list_idx < subgraph_vertices_count) {
          size_t vertex = frame_list_idx;
          frame_list_idx++;
          if (!frame.is_neighbour(vertex)) {
            continue;
          }

          int num_neighbours;
          if (lastLevel) {
            num_neighbours = frame.intersect(adj.matrix[vertex]);
          } else {
            num_neighbours = frame.intersect(adj.matrix[vertex], &stack[level - 1]);
          }
          localRes[level] += num_neighbours;
          localRes[level] %= MOD;

          if (!lastLevel && __ballot_sync(FULL_MASK, num_neighbours)) {
            // At least one neighbour found.
            recurse = true;
            break;
          }
        }

        if (recurse) {
          stackListIdx[level - 1] = 0;
          level++;
        } else {
          level--;
        }
      }
    }
  }

  // Reduce local answers to a block answer.
  __shared__ unsigned long long blockRes[THREADS / 2][MAX_K - 2];

  for (unsigned i = threadTotalIdx() % 2; i < MAX_K - 2; i += 2) {
    blockRes[threadTotalIdx() / 2][i] = 0;
  }
  __syncthreads();

  reduceBlockRes(blockRes, localRes, k);

  if (isBlockLeader) {
    reduceRes(blockRes, localRes, out, k);
  }
}

void handle_error(hipError_t err) {
  if (err != hipSuccess) {
    throw std::runtime_error("CUDA operations resulted in failure, code: " + std::to_string(err));
  }
}

std::vector<unsigned long long> find_cliques(unsigned long k, const CSR &csr) {
  std::vector<unsigned long long> ret(k);
  if (k > 1) {
    unsigned long long *out;
    size_t *row_ptr;
    size_t *col_ind;

    handle_error(hipMalloc(&out, (k - 1) * sizeof(unsigned long long))); // TODO scale out down.
    handle_error(hipMalloc(&row_ptr, csr.row_ptr.size() * sizeof(size_t)));
    handle_error(hipMalloc(&col_ind, csr.col_ind.size() * sizeof(size_t)));

    handle_error(hipMemcpy(row_ptr, csr.row_ptr.data(), csr.row_ptr.size() * sizeof(size_t), hipMemcpyHostToDevice));
    handle_error(hipMemcpy(col_ind, csr.col_ind.data(), csr.col_ind.size() * sizeof(size_t), hipMemcpyHostToDevice));

    handle_error(hipMemset(out, 0, (k - 1) * sizeof(unsigned long long)));

    unsigned long long *counter;
    handle_error(hipMalloc(&counter, sizeof(unsigned long long)));
    handle_error(hipMemset(counter, 0, sizeof(unsigned long long)));

    AdjMatrix *matrices;
    handle_error(hipMalloc(&matrices, sizeof(AdjMatrix) * BLOCKS));
    handle_error(hipMemset(matrices, 0, sizeof(AdjMatrix) * BLOCKS));

    dim3 threadsDim(WARP_SIZE, GROUPS_PER_BLOCK);
    clique_kernel<<<BLOCKS, threadsDim>>>(row_ptr,
                                          csr.row_ptr.size(),
                                          col_ind,
                                          k,
                                          out,
                                          counter,
                                          matrices);

    handle_error(hipPeekAtLastError());
    handle_error(hipDeviceSynchronize());

    handle_error(hipMemcpy(ret.data() + 1, out, (k - 1) * sizeof(unsigned long long), hipMemcpyDeviceToHost));

    handle_error(hipFree(matrices));
    handle_error(hipFree(out));
    handle_error(hipFree(row_ptr));
    handle_error(hipFree(col_ind));
    handle_error(hipFree(counter));

    for (unsigned long long &el : ret) {
      el %= MOD;
    }
  }

  ret[0] = csr.row_ptr.size() - 1; // Number of 1-cliques is the number of vertices.

  return ret;
}

}

typedef uint32_t vid;

struct UndirectedEdge {
  vid a;
  vid b;

  UndirectedEdge(vid _a, vid _b) : a(_a), b(_b) {}
};

struct Edge {
  vid from;
  vid to;

  Edge(vid _from, vid _to) : from(_from), to(_to) {}
};

struct Graph {
  //    Vertices are numbered by consecutive numbers starting with 1.
  std::vector<UndirectedEdge> edges;
  vid vLimit;

  std::vector<Edge> toOrientedEdges() {
    std::unordered_map<vid, size_t> neighbours_count;
    for (const auto &e : edges) {
      neighbours_count[e.a]++;
      neighbours_count[e.b]++;
    }

    std::vector<Edge> orientedEdges;
    for (const auto &e : edges) {
      if (neighbours_count[e.a] < neighbours_count[e.b]) {
        orientedEdges.emplace_back(e.a, e.b);
      } else if (neighbours_count[e.a] > neighbours_count[e.b]) {
        orientedEdges.emplace_back(e.b, e.a);
      } else {
        if (e.a < e.b) {
          orientedEdges.emplace_back(e.a, e.b);
        } else {
          orientedEdges.emplace_back(e.b, e.a);
        }
      }
    }

    return orientedEdges;
  }

  CSR toCSR() {
    auto orientedEdges = toOrientedEdges();
    std::sort(orientedEdges.begin(),
              orientedEdges.end(),
              [](const Edge &a, const Edge &b) {
                if (a.from == b.from) {
                  return a.to < b.to;
                }
                return a.from < b.from;
              });
    CSR csr{};
    csr.row_ptr.reserve(vLimit + 1);

    size_t curr_ptr = 0;
    for (const auto &e : orientedEdges) {
      while (csr.row_ptr.size() <= e.from) {
        csr.row_ptr.push_back(curr_ptr);
      }
      curr_ptr++;
      csr.col_ind.push_back(e.to);
    }
    while (csr.row_ptr.size() <= vLimit) {
      csr.row_ptr.push_back(curr_ptr);
    }

    return csr;
  }
};

Graph read_graph(std::ifstream &in) {
  Graph g{};
  std::unordered_map<vid, vid> mapping;
  vid currVid = 0;
  vid a, b;
  while (in >> a >> b) {
    auto pA = mapping.find(a);
    if (pA == mapping.end()) {
      pA = mapping.insert({a, currVid}).first;
      currVid++;
    }

    auto pB = mapping.find(b);
    if (pB == mapping.end()) {
      pB = mapping.insert({b, currVid}).first;
      currVid++;
    }

    g.edges.emplace_back(pA->second, pB->second);
  }

  g.vLimit = currVid;

  return g;
}

void kclique(std::ifstream &in, unsigned long k, std::ofstream &out) {
  Graph g = read_graph(in);

  auto ret = gpu::find_cliques(k, g.toCSR());

  for (size_t i = 0; i < k; i++) {
    out << ret[i] << " ";
  }
  out << std::endl;
}

int main(int argc, char **argv) {
  if (argc != 4) {
    std::cerr << "Usage: " << argv[0] << " <Graph input file> <k value> <output file>" << std::endl;
    return 1;
  }
  std::ifstream in{argv[1]};
  if (!in.is_open()) {
    std::cerr << "Input file: " << argv[1] << " doesn't exist." << std::endl;
    return 1;
  }
  char *endptr;
  unsigned long k = strtoul(argv[2], &endptr, 0);
  if (*endptr != '\0') {
    std::cerr << "Invalid k value." << std::endl;
    return 1;
  }
  if (k == 0 || k > 12) {
    std::cerr << "K value must be in range (0, 12]." << std::endl;
    return 1;
  }
  std::ofstream out{argv[3]};
  if (!out.is_open()) {
    std::cerr << "Error opening output file: " << argv[3] << std::endl;
    return 1;
  }

  kclique(in, k, out);

  return 0;
}
