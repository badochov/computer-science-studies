#include "hip/hip_runtime.h"
/**
 * stencil.cu: a simple 1d stencil on GPU and on CPU
 * 
 * Implement the basic stencil and make sure it works correctly.
 * Then, play with the code
 * - Experiment with block sizes, various RADIUSes and NUM_ELEMENTS.
 * - Measure the memory transfer time, estimate the effective memory bandwidth.
 * - Estimate FLOPS (floating point operations per second)
 * - Switch from float to double: how the performance changes?
*/

#include <time.h>
#include <stdio.h>
#include <cassert>

#include <algorithm>

#define RADIUS 3
#define NUM_ELEMENTS int(1e8)
#define THREADS_PER_BLOCK int(1024)

#define s_t double

#define BYTES (NUM_ELEMENTS * sizeof(s_t))

static void handleError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess)
  {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define cudaCheck(err) (handleError(err, __FILE__, __LINE__))

__global__ void stencil_1d(s_t *in, s_t *out)
{
  int idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  if (idx >= NUM_ELEMENTS)
  {
    return;
  }

  out[idx] = 0;
  for (int j = max(0, idx - RADIUS); j < min(idx + RADIUS + 1, NUM_ELEMENTS); j++)
  {
    out[idx] += in[j];
  }
}

void cpu_stencil_1d(s_t *in, s_t *out)
{
  for (int i = 0; i < NUM_ELEMENTS; i++)
  {
    out[i] = 0;
    for (int j = max(0, i - RADIUS); j < min(i + RADIUS + 1, NUM_ELEMENTS); j++)
    {
      out[i] += in[j];
    }
  }
}

int main()
{
  s_t *in = (s_t *)calloc(NUM_ELEMENTS, sizeof(s_t));
  s_t *out = (s_t *)calloc(NUM_ELEMENTS, sizeof(s_t));
  s_t *outd = (s_t *)calloc(NUM_ELEMENTS, sizeof(s_t));
  s_t *devIn, *devOut;

  for (int i = 0; i < NUM_ELEMENTS; i++)
  {
    in[i] = i - NUM_ELEMENTS / 2;
  }

  hipEvent_t start, stop;
  hipEvent_t startTran, stopTran;
  hipEvent_t startTranFrom, stopTranFrom;
  cudaCheck(hipEventCreate(&start));
  cudaCheck(hipEventCreate(&stop));
  cudaCheck(hipEventCreate(&startTran));
  cudaCheck(hipEventCreate(&stopTran));
  cudaCheck(hipEventCreate(&startTranFrom));
  cudaCheck(hipEventCreate(&stopTranFrom));
  cudaCheck(hipEventRecord(start, 0));

  cudaCheck(hipMalloc((void **)&devIn, NUM_ELEMENTS * sizeof(s_t)));
  cudaCheck(hipMalloc((void **)&devOut, NUM_ELEMENTS * sizeof(s_t)));

  cudaCheck(hipEventRecord(startTran, 0));
  cudaCheck(hipMemcpy(devIn, in, NUM_ELEMENTS * sizeof(s_t), hipMemcpyHostToDevice));
  cudaCheck(hipEventRecord(stopTran, 0));
  cudaCheck(hipEventSynchronize(stopTran));

  stencil_1d<<<NUM_ELEMENTS / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK>>>(devIn, devOut);
  cudaCheck(hipPeekAtLastError());

  cudaCheck(hipEventRecord(startTranFrom, 0));
  
  cudaCheck(hipMemcpy(outd, devOut, NUM_ELEMENTS * sizeof(s_t), hipMemcpyDeviceToHost));
  cudaCheck(hipEventRecord(stopTranFrom, 0));
  cudaCheck(hipEventSynchronize(stopTranFrom));

  cudaCheck(hipEventRecord(stop, 0));
  cudaCheck(hipEventSynchronize(stop));

  float gpuTranToElapsedTime;
  hipEventElapsedTime(&gpuTranToElapsedTime, startTran, stopTran);
  printf("Total GPU transfer to time:  %.4f ms, %.4f GB/s\n", gpuTranToElapsedTime, BYTES / gpuTranToElapsedTime / 1e6);
  cudaCheck(hipEventDestroy(startTran));
  cudaCheck(hipEventDestroy(stopTran));

  float gpuTranFromElapsedTime;
  hipEventElapsedTime(&gpuTranFromElapsedTime, startTranFrom, stopTranFrom);
  printf("Total GPU transfer from time:  %.4f ms, %.4f GB/s\n", gpuTranFromElapsedTime, BYTES / gpuTranFromElapsedTime /1e6 );
  cudaCheck(hipEventDestroy(startTranFrom));
  cudaCheck(hipEventDestroy(stopTranFrom));


  float gpuElapsedTime;
  hipEventElapsedTime(&gpuElapsedTime, start, stop);
  printf("Total GPU execution time:  %.4f ms, %.4f GFLOPS\n", gpuElapsedTime, NUM_ELEMENTS * RADIUS / (gpuElapsedTime - gpuTranFromElapsedTime - gpuTranToElapsedTime) / 1e6);
  cudaCheck(hipEventDestroy(start));
  cudaCheck(hipEventDestroy(stop));

  cudaCheck(hipFree(devIn));
  cudaCheck(hipFree(devOut));

  struct timespec cpu_start, cpu_stop;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);

  cpu_stencil_1d(in, out);

  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
  double cpuElapsedTime = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
  printf("CPU execution time:  %.4f ms\n", cpuElapsedTime);

  for (int i = 0; i < NUM_ELEMENTS; i++)
  {
    if (abs(outd[i] - out[i]) > 1e-6)
    {
      printf("%d %f %f\n", i, outd[i], out[i]);
      return 1;
    }
  }

  free(in);
  free(out);
  free(outd);

  return 0;
}
