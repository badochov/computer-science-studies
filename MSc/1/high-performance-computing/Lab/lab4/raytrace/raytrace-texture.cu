#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common/errors.h"
#include "common/cpu_bitmap.h"

#define DIM 1024
#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f
#define SPHERES 100

texture<float, 1> t_red, t_green, t_blue, t_radius, t_x, t_y, t_z;

__device__ float hit(float red, float green, float blue, float radius, float x, float y, float z, float bitmapX, float bitmapY, float *colorFalloff)
{
	float distX = bitmapX - x;
	float distY = bitmapY - y;

	if (distX * distX + distY * distY < radius * radius)
	{
		float distZ = sqrtf(radius * radius - distX * distX - distY * distY);
		*colorFalloff = distZ / sqrtf(radius * radius);
		return distZ + z;
	}

	return -INF;
}

__global__ void kernel(unsigned char *bitmap)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	// tex

	float bitmapX = (x - DIM / 2);
	float bitmapY = (y - DIM / 2);

	float red = 0, green = 0, blue = 0;
	float maxDepth = -INF;

	for (int i = 0; i < SPHERES; i++)
	{
		float colorFalloff;
		float depth = hit(tex1D(t_red, i), tex1D(t_green, i), tex1D(t_blue, i), tex1D(t_radius, i), tex1D(t_x, i), tex1D(t_y, i), tex1D(t_z, i), bitmapX, bitmapY, &colorFalloff);

		if (depth > maxDepth)
		{
			red = tex1D(t_red, i) * colorFalloff;
			green = tex1D(t_green, i) * colorFalloff;
			blue = tex1D(t_blue, i) * colorFalloff;
			maxDepth = depth;
		}
	}

	bitmap[offset * 4 + 0] = (int)(red * 255);
	bitmap[offset * 4 + 1] = (int)(green * 255);
	bitmap[offset * 4 + 2] = (int)(blue * 255);
	bitmap[offset * 4 + 3] = 255;
}

struct DataBlock
{
	unsigned char *hostBitmap;
};

int main(void)
{
	DataBlock data;
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	CPUBitmap bitmap(DIM, DIM, &data);
	unsigned char *devBitmap;

	HANDLE_ERROR(hipMalloc((void **)&devBitmap, bitmap.image_size()));

	float host_red[SPHERES];
	float host_blue[SPHERES];
	float host_green[SPHERES];
	float host_radius[SPHERES];
	float host_x[SPHERES];
	float host_y[SPHERES];
	float host_z[SPHERES];

	float *d_red, *d_green, *d_blue, *d_radius, *d_x, *d_y, *d_z;
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	hipMalloc((void **)&d_red, SPHERES * sizeof(float));
	hipMalloc((void **)&d_green, SPHERES * sizeof(float));
	hipMalloc((void **)&d_blue, SPHERES * sizeof(float));
	hipMalloc((void **)&d_radius, SPHERES * sizeof(float));
	hipMalloc((void **)&d_x, SPHERES * sizeof(float));
	hipMalloc((void **)&d_y, SPHERES * sizeof(float));
	hipMalloc((void **)&d_z, SPHERES * sizeof(float));

	hipBindTexture(0, t_red, d_red, desc, SPHERES * sizeof(float));
	hipBindTexture(0, t_green, d_green, desc, SPHERES * sizeof(float));
	hipBindTexture(0, t_blue, d_blue, desc, SPHERES * sizeof(float));
	hipBindTexture(0, t_radius, d_radius, desc, SPHERES * sizeof(float));
	hipBindTexture(0, t_x, d_x, desc, SPHERES * sizeof(float));
	hipBindTexture(0, t_y, d_y, desc, SPHERES * sizeof(float));
	hipBindTexture(0, t_z, d_z, desc, SPHERES * sizeof(float));

	for (int i = 0; i < SPHERES; i++)
	{
		host_red[i] = rnd(1.0f);
		host_green[i] = rnd(1.0f);
		host_blue[i] = rnd(1.0f);
		host_x[i] = rnd(1000.0f) - 500;
		host_y[i] = rnd(1000.0f) - 500;
		host_z[i] = rnd(1000.0f) - 500;
		host_radius[i] = rnd(100.0f) + 20;
	}

	hipMemcpy(d_red, host_red, SPHERES * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_green, host_green, SPHERES * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_blue, host_blue, SPHERES * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_radius, host_radius, SPHERES * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, host_x, SPHERES * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, host_y, SPHERES * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_z, host_z, SPHERES * sizeof(float), hipMemcpyHostToDevice);

	dim3 grids(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	kernel<<<grids, threads>>>(devBitmap);

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Time to generate: %3.1f ms\n", elapsedTime);

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), devBitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	bitmap.dump_ppm("image-texture.ppm");

	hipUnbindTexture(t_red);
	hipUnbindTexture(t_green);
	hipUnbindTexture(t_blue);
	hipUnbindTexture(t_x);
	hipUnbindTexture(t_y);
	hipUnbindTexture(t_z);
	hipUnbindTexture(t_radius);

	hipFree(devBitmap);
}

// Global
